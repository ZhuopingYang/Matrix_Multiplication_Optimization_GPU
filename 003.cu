/**
 * shared memory cache-blocking, 1927.7 GOPs
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <string.h>

#define BLOCK_SIZE 32

#define CEIL_DIV(x, y) ((int) ((x + 0.5) / y) )

__global__ void matrix_multiplication_gpu(const float *A, const float *B, float *C, int N){
    
    const uint x = blockIdx.x * BLOCK_SIZE + threadIdx.x / BLOCK_SIZE;
    const uint y = blockIdx.y * BLOCK_SIZE + threadIdx.x % BLOCK_SIZE;

    const uint sub_row = threadIdx.x / BLOCK_SIZE;
    const uint sub_col = threadIdx.x % BLOCK_SIZE;

    __shared__ float buf_a[BLOCK_SIZE * BLOCK_SIZE];
    __shared__ float buf_b[BLOCK_SIZE * BLOCK_SIZE];

    if(x < N && y < N){
        float tmp = 0;
        for(int i = 0; i < N; i += BLOCK_SIZE){
            // copy data from global memory to shared memory

            buf_a[sub_row * BLOCK_SIZE + sub_col] = A[(blockIdx.x * BLOCK_SIZE + sub_row) * N + i + sub_col];
            buf_b[sub_row * BLOCK_SIZE + sub_col] = B[(i + sub_row) * N + (blockIdx.y * BLOCK_SIZE + sub_col)];

            __syncthreads();

            for(int j = 0; j < BLOCK_SIZE; ++j){
                tmp += buf_a[sub_row * BLOCK_SIZE + j] * buf_b[(j) * BLOCK_SIZE + sub_col];
            }

            __syncthreads();

        }
        C[x * N + y] = tmp;
    }
}

void matrix_multiplication_cpu(const float *A, const float *B, float *C, int N){
    for(int i = 0; i < N; ++i){
        for(int k = 0; k < N; ++k){
            for(int j = 0; j < N; ++j){
                C[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
}

void matrix_init(float * M, int size){
    for(int i = 0; i < size; ++i){
        M[i] = ((float) rand() * 3) / RAND_MAX;
    }
}

void verifygpu(float * C, float * D, int size){
    for(int i = 0; i < size; ++i){
        // printf("%d %f %f\r\n", i, C[i], D[i]);
        if(fabs(C[i] - D[i]) > 1e-3){
            printf("%d %f %f mismatch\r\n", i, C[i], D[i]);
            printf("Test Failed!\r\n");
            return;
        }
    }
    printf("Test Passed!\r\n");
}

int main(int argc, char *argv[]){

    int size = 1024;
    int repeat = 1;
    int verify = 1;

    if(argc > 3){
        size = atoi(argv[1]);
        repeat = atoi(argv[2]);
        verify = atoi(argv[3]);
    }

    float * A = (float *) malloc(sizeof(float) * size * size);
    float * B = (float *) malloc(sizeof(float) * size * size);
    float * C = (float *) malloc(sizeof(float) * size * size);
    float * C_cuda = (float *) malloc(sizeof(float) * size * size);
    srand(time(0));
    matrix_init(A, size * size);
    matrix_init(B, size * size);
    memset(C, 0, sizeof(float) * size * size);

    dim3 gridDim(CEIL_DIV(size, BLOCK_SIZE), CEIL_DIV(size, BLOCK_SIZE), 1);
    dim3 blockDim(BLOCK_SIZE * BLOCK_SIZE);
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size * size * sizeof(float));
    hipMalloc(&d_B, size * size * sizeof(float));
    hipMalloc(&d_C, size * size * sizeof(float));

    hipMemcpy(d_A, A, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * size * sizeof(float), hipMemcpyHostToDevice);

    clock_t start, end;
    double time_elapse = 0;
    double perf = 0;

    if(verify){
        start = clock();
        matrix_multiplication_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, size);
        hipDeviceSynchronize();
        end = clock();
        time_elapse = ((double) end - start) / CLOCKS_PER_SEC;
        perf = (double) size * size * size * 2 / time_elapse / 1e9;
        printf("gpu execution time: %.5f sec, performance: %.3f GOPs\r\n", time_elapse, perf);
        hipMemcpy(C_cuda, d_C, size * size * sizeof(float), hipMemcpyDeviceToHost);
        matrix_multiplication_cpu(A, B, C, size);
        verifygpu(C, C_cuda, size * size);
    }else{
        start = clock();

        for(int rep = 0; rep < repeat; ++rep){
            matrix_multiplication_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, size);
            hipDeviceSynchronize();
        }

        end = clock();
        time_elapse = ((double) end - start) / CLOCKS_PER_SEC;
        perf = (double) size * size * size * 2. * repeat / time_elapse / 1e9;
        
        hipError_t err{hipGetLastError()};
        if (err != hipSuccess){
            printf("error: %s\r\n", hipGetErrorString(err));
            // We don't exit when we encounter CUDA errors in this example.
            // std::exit(EXIT_FAILURE);
        }else{
            printf("gpu execution time: %.5f sec, performance: %.3f GOPs\r\n", time_elapse, perf);
        }

    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);
    free(C_cuda);


    return 0;
}